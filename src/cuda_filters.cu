#include "hip/hip_runtime.h"
extern "C" {
#include "cuda_filters.h"
}

#define BLOCK_DIM 512
__global__ void
gray_kernel( pixel * p, int max){
    int i;
    i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < max){
        int moy ;

        moy = (p[i].r + p[i].g + p[i].b)/3 ;
        if ( moy < 0 ) moy = 0 ;
        if ( moy > 255 ) moy = 255 ;

        p[i].r = moy ;
        p[i].g = moy ;
        p[i].b = moy ;
    }
}

int grid_dim(int data_size, int block_dim){
    return data_size/block_dim + 1;
}

void gray_filter_cuda(pixel* p, int width, int height)
{
    pixel * d_p;
    hipMalloc((void **)&d_p, width * height * sizeof(pixel));
    hipMemcpy(d_p, p, width * height * sizeof(pixel), hipMemcpyHostToDevice);
    gray_kernel<<<grid_dim(width*height, BLOCK_DIM),BLOCK_DIM>>>(d_p, width * height);

    hipMemcpy(p, d_p, width * height * sizeof(pixel), hipMemcpyDeviceToHost);
}
